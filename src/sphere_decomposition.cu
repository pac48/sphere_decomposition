#include "hip/hip_runtime.h"
#include "sphere_decomposition.hpp"
#include "memory"
#include "chrono"
#include "iostream"

namespace sphere_decomposition {
  template<typename T>
  struct BufferGPU {
    T *buffer;
    size_t size;

    explicit BufferGPU(size_t size_in) : size{size_in} {
      hipMalloc(&buffer, size * sizeof(T));
      hipMemset(&buffer, 0, size * sizeof(T));
    }

    BufferGPU(const BufferGPU &other) {
      size = other.size;
      hipMalloc(&buffer, size * sizeof(T));
      hipMemcpy(&buffer, &other, size * sizeof(T), hipMemcpyKind::hipMemcpyDeviceToDevice);
    }

    BufferGPU &operator=(const BufferGPU &other) {
      hipFree(buffer);
      size = other.size;
      hipMalloc(&buffer, size * sizeof(T));
      hipMemcpy(&buffer, &other, size * sizeof(T), hipMemcpyKind::hipMemcpyDeviceToDevice);
      return *this;
    }

    ~BufferGPU() {
      hipFree(buffer);
    }

    std::vector<T> toCPU() {
      std::vector<T> out(size);
      hipMemcpy(out.data(), buffer, size * sizeof(T), hipMemcpyKind::hipMemcpyDeviceToHost);

      return out;
    }

  };

  std::shared_ptr<BufferGPU<unsigned char>> gpu_pixel_buffer = nullptr;
  std::shared_ptr<BufferGPU<double>> gpu_triangle_buffer = nullptr;


  struct Vertex {
    double x;
    double y;
    double z;
  };

  typedef Vertex Vector;

  inline __device__ float dot_product(const Vertex &a, const Vertex &b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
  }

  inline __device__ Vertex vec_minus(const Vertex &vert1, const Vertex &vert2) {
    Vertex out;
    out.x = vert1.x - vert2.x,
    out.y = vert1.y - vert2.y,
    out.z = vert1.z - vert2.z;
    return out;
  }

  struct Triangle {
    Vertex v1;
    Vertex v2;
    Vertex v3;
  };

  inline __device__ void bary_centric(Vertex a, Vertex b, Vertex c, Vertex p, float &u, float &v, float &w) {
    Vertex v0 = vec_minus(b, a);
    Vertex v1 = vec_minus(c, a);
    Vertex v2 = vec_minus(p, a);
    float d00 = dot_product(v0, v0);
    float d01 = dot_product(v0, v1);
    float d11 = dot_product(v1, v1);
    float d20 = dot_product(v2, v0);
    float d21 = dot_product(v2, v1);
    float denom = d00 * d11 - d01 * d01;
    v = (d11 * d20 - d01 * d21) / denom;
    w = (d00 * d21 - d01 * d20) / denom;
    u = 1.0f - v - w;
  }

  inline Vector __device__ cross(const Vector &a, const Vector &b) {
    Vector out;
    out.x = a.y * b.z - a.z * b.y;
    out.y = a.z * b.x - b.z * a.x;
    out.z = a.x * b.y - a.y * b.x;

    return out;
  }

  inline float __device__ cross_Z(const Vector &a, const Vector &b) {
    return a.x * b.y - a.y * b.x;

  }

  __device__ int is_point_in_triangle(const Vertex &a, const Vertex &b, const Vertex &c, const Vertex &p) {
    // if z component of cross product is positive, then the point is inside for convex mesh
    float val1 = cross_Z(vec_minus(b, a), vec_minus(p, a));
    float val2 = cross_Z(vec_minus(c, b), vec_minus(p, b));
    float val3 = cross_Z(vec_minus(a, c), vec_minus(p, c));
    return (val1 < 0 && val2 < 0 && val3 < 0);
  }

  __device__ void project_triangle(float fx, float fy, Triangle &triangle) {
    triangle.v1.x = fx * triangle.v1.x / triangle.v1.z;
    triangle.v1.y = fy * triangle.v1.y / triangle.v1.z;
    triangle.v1.z = 1;

    triangle.v2.x = fx * triangle.v2.x / triangle.v2.z;
    triangle.v2.y = fy * triangle.v2.y / triangle.v2.z;
    triangle.v2.z = 1;

    triangle.v3.x = fx * triangle.v3.x / triangle.v3.z;
    triangle.v3.y = fy * triangle.v3.y / triangle.v3.z;
    triangle.v3.z = 1;
  }

  constexpr double MAX_DEPTH = 1E99;

  __global__ void render_kernel(const double *triangles, size_t size, unsigned char *image, size_t image_size,
                                float fx, float fy, unsigned int res_x, unsigned int res_y) {
    // Get the index of the current thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure we don't go out of bounds
    if (idx < image_size / 4) {
      unsigned int ind_x = idx % res_x;
      unsigned int ind_y = idx / res_x;
      double depth = MAX_DEPTH;
      image[idx * 4] = 0;
      image[idx * 4 + 1] = 0;
      image[idx * 4 + 2] = 0;
      image[idx * 4 + 3] = 255;


      for (size_t ind = 0; ind < size; ind += 9) {
        const Triangle &triangle = *(Triangle *) &triangles[ind];
        Triangle triangle2d = *(Triangle *) &triangles[ind];
        // TODO does this need to be normalized?
        project_triangle(fx, fy, triangle2d);
        Vertex point;
        point.x = 2.0 * (ind_x - res_x / 2.0) / res_x;
        point.y = 2.0 * (ind_y - res_y / 2.0) / res_y;
        point.z = 1.0;
        if (is_point_in_triangle(triangle2d.v1, triangle2d.v2, triangle2d.v3, point) == true) {
          // ax + by + cz + d = 0;
          // z = -(ax + by + d)/c;
//          double new_depth = -(normal.x * point.x + normal.y * point.y + intercept) / normal.z;
//          double intercept = -(normal.x * triangle.v1.x + normal.y * triangle.v1.y + normal.z * triangle.v1.z);
          float u, v, w;
          bary_centric(triangle2d.v1, triangle2d.v2, triangle2d.v3, point, u, v, w);
          point.x = triangle.v1.x * u + triangle.v2.x * v + triangle.v3.x * w;
          point.y = triangle.v1.y * u + triangle.v2.y * v + triangle.v3.y * w;
          point.z = triangle.v1.z * u + triangle.v2.z * v + triangle.v3.z * w;

          // now normalize norm vector
          Vector normal = cross(vec_minus(triangle.v2, triangle.v1), vec_minus(triangle.v3, triangle.v1));
          double length = sqrt(normal.x * normal.x + normal.y * normal.y + normal.z * normal.z);
          normal.x = normal.x / length;
          normal.y = normal.y / length;
          normal.z = normal.z / length;

          if (point.z < depth) { // && normal.z > 0 && normal.z < 0
//            printf("depth: %f\n", new_depth);
//            printf("thread id: %d\n", idx);
            depth = point.z;
            image[idx * 4] = (double) -normal.z * 200;
            image[idx * 4 + 1] = (double) -normal.z * 200;
            image[idx * 4 + 2] = (double) -normal.z * 200;
            image[idx * 4 + 3] = 255;
          }
        }
      }
    }
  }


  std::vector<unsigned char> render(float fx, float fy, unsigned int res_x, unsigned int res_y, CArray triangles) {
    if (gpu_pixel_buffer == nullptr || res_x * res_y * 4 > gpu_pixel_buffer->size) {
      gpu_pixel_buffer = std::make_shared<BufferGPU<unsigned char>>(res_x * res_y * 4);
    }
    if (gpu_triangle_buffer == nullptr || triangles.size > gpu_triangle_buffer->size) {
      gpu_triangle_buffer = std::make_shared<BufferGPU<double>>(triangles.size);
    }
    hipMemcpy(gpu_triangle_buffer->buffer, triangles.data, triangles.size * sizeof(double),
               hipMemcpyKind::hipMemcpyHostToDevice);
    dim3 threadsPerBlock(256);
    dim3 numBlocks((gpu_pixel_buffer->size / 4 + threadsPerBlock.x - 1) / threadsPerBlock.x);
    hipStream_t stream;
    hipStreamCreate(&stream);
    auto start = std::chrono::high_resolution_clock::now();
    render_kernel<<<numBlocks, threadsPerBlock, 0, stream>>>(
        gpu_triangle_buffer->buffer, gpu_triangle_buffer->size, gpu_pixel_buffer->buffer, gpu_pixel_buffer->size, fx,
        fy, res_x, res_y);
    hipStreamSynchronize(stream);
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    std::cout << "Time taken by render: " << (double) duration.count() << " microseconds" << std::endl;

    start = std::chrono::high_resolution_clock::now();
    auto out = gpu_pixel_buffer->toCPU();
    stop = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    std::cout << "Time taken by copy data: " << (double) duration.count() << " microseconds" << std::endl;

    hipStreamDestroy(stream);

    return out;
  }

}
